#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define RESET "\033[0m"

#define maxThreadsPerBlock 64 //8*8

void makeAdjacency();
void floydWarshallMethod();
void methods(int f);
void test(float *A_dev);
void printArray(float *array);

__global__ void kernel1(int k, int n, float *A);
__global__ void kernel2(int k, int n, float *A);
__global__ void kernel3(int k, int n, float *A, int cellsPerThread);

int n, w;		
float p;
float *D; 		/* D:distance (for the Floyd Warshall algorithm) */
float *A; 		/* A:array (for methods 1, 2, 3 as mentioned in the exercise) */
float *A_result1, *A_result2, *A_result3;	/* arrays for saving results in methods 1, 2, 3 */

float seq_time;
struct timeval startwtime, endwtime;
	hipEvent_t start, stop;

//--------------------------------------------------------------------------------------------------------------

int main(int argc, char** argv)
{

	if (argc!=4){
		printf("Enter: ./<name> n p w\n");
		exit(3);
	}

	n = 1<<atoi(argv[1]);
	p = atof(argv[2]);
	w = atoi(argv[3]);
    
    time_t t;
    srand((unsigned)time(&t));

    D = (float *)malloc(n*n*sizeof(float)); 
    A = (float *)malloc(n*n*sizeof(float)); 

	printf("n = %d \np = %.2f \nw = %d\n", n, p, w);
	

	makeAdjacency(); 	/* initialize the graph */
	printArray(A);

	floydWarshallMethod();
	printf("Floyd Warshall time:\t%f\n", seq_time);
	printArray(D);

	methods(1);
	printf("CUDA Method 1 time:\t%f -> ", seq_time);
	test(A_result1);
	printArray(A_result1);
	free(A_result1);

	methods(2);
	printf("CUDA Method 2 time:\t%f -> ", seq_time);
    test(A_result2);
    printArray(A_result2);
    free(A_result2);

	methods(3);
	printf("CUDA Method 3 time:\t%f -> ", seq_time);
    test(A_result3);
    printArray(A_result3);
	free(A_result3);

	free(D);
	free(A);
    return 1;
}

//--------------------------------------------------------------------------------------------------------------

/* create the array that represents the graph */
void makeAdjacency()
{
	int i, j;
	float randValue;

	for (i=0; i<n; i++){
		for (j=0; j<n; j++){
			randValue = (((double)rand()+1)/((double)RAND_MAX+1)); /* Get a random float value from 0 to 1 */
			if (randValue>p)
				A[i*n+j] = INFINITY;
			else 
				A[i*n+j] = randValue * w;
		}
        A[i*n+i] = 0;
	}
}

//--------------------------------------------------------------------------------------------------------------

/* print array for personal check (only for small n) */
void printArray(float *array) {
    int i;
    
    if (n>3*3) return; 

    for (i=0; i<n*n; i++) {
    	if (i % n == 0) printf("\n"); /* change line */
    	printf("%f  ",array[i]);
    }

    printf("\n");	
}

//--------------------------------------------------------------------------------------------------------------

/* floyd Warshall Method based on this: http://www.programming-algorithms.net/article/45708/Floyd-Warshall-algorithm */
void floydWarshallMethod()
{
	int i, j, k;

	/* copy arrays D <- A */
	for (i=0; i<n; i++){
		for (j=0; j<n; j++){
			D[i*n+j] = A[i*n+j];
		}
		D[i*n+i] = 0;
	}

    gettimeofday (&startwtime, NULL);

	for (k=0; k<n; k++){
		for (i=0; i<n; i++){
			for (j=0; j<n; j++){
                if (D[i*n+j] > D[i*n+k] + D[k*n+j]){
                    D[i*n+j] = D[i*n+k] + D[k*n+j];
                }
			}
		}
	}
	gettimeofday (&endwtime, NULL);
    seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
}


//--------------------------------------------------------------------------------------------------------------

/* calling kernel functions for methods 1, 2, 3 and calculating the execution time */
/* where f = 1, 2, 3 for methods 1, 2 or 3 respectively */
void methods(int f)
{
	float *A_dev;															/* gpu array */
	int k;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void **)&A_dev, n*n*sizeof(float)); 						/* allocate the memory in GPU */ 		

    /****** 1st Method ******/
	if (f==1){
		A_result1 = (float *) malloc (n*n*sizeof(float));
		hipEventRecord(start);												/* start time counter */
		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */
		

		if (n*n<=maxThreadsPerBlock){ 										/* 64 threads (max) per block */
	     	dim3 threadsPerBlock(n, n);
	    	for (k=0; k<n; k++)
				kernel1<<<1, threadsPerBlock>>>(k, n, A_dev);	
		}
		else{			/* if threads per block > 64 */
			dim3 threadsPerBlock(8,8);
			dim3 blocksPerGrid(n/threadsPerBlock.x, n/threadsPerBlock.y);
			for (k=0; k<n; k++)
				kernel1<<<blocksPerGrid, threadsPerBlock>>>(k, n, A_dev);
		}
	   
	   hipMemcpy(A_result1, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost); /* get results back to CPU */
	}


	/****** 2nd Method ******/
	/* same as 1st method but this time we are calling different kernel function that uses shared mem */
	else if (f==2){
		A_result2 = (float *) malloc (n*n*sizeof(float));
		hipEventRecord(start);												/* start time counter */
		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */


		if (n*n<=maxThreadsPerBlock){ 										/* 64 threads (max) per block */
	     	dim3 threadsPerBlock(n, n);
	    	for (k=0; k<n; k++)
	    		kernel2<<<1, threadsPerBlock>>>(k, n, A_dev);
		}
		else{
			dim3 threadsPerBlock(8,8);
			dim3 blocksPerGrid(n/threadsPerBlock.x, n/threadsPerBlock.y);
			for (k=0; k<n; k++)
				kernel2<<<blocksPerGrid, threadsPerBlock>>>(k, n, A_dev);
		}
	   
	   	hipMemcpy(A_result2, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}


	/****** 3rd Method ******/
	/* same as 2nd method but based on 4 cells per thread */
	else if (f==3){
		A_result3 = (float *) malloc (n*n*sizeof(float));
		hipEventRecord(start);												/* start time counter */
		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */
		int cellsPerThread = 4;												/* number of cells per thread */
		
		if (n*n<=maxThreadsPerBlock * cellsPerThread){ 
   	    	dim3 threadsPerBlock(n/cellsPerThread, n/cellsPerThread);
	    	for (k=0; k<n; k++)
	    		kernel3<<<1, threadsPerBlock>>>(n, k, A_dev, cellsPerThread);
		}
		else{
			dim3 threadsPerBlock(8,8);
    		dim3 blocksPerGrid(n/(threadsPerBlock.x * cellsPerThread), n/(threadsPerBlock.y * cellsPerThread));
			for (k=0; k<n; k++)
				kernel3<<<blocksPerGrid, threadsPerBlock>>>(k, n, A_dev, cellsPerThread);
		}
	    
	    hipMemcpy(A_result3, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}


    hipEventRecord(stop);								/* stop time countdown*/
    hipEventSynchronize(stop);
    hipEventElapsedTime(&seq_time, start, stop);
    seq_time=seq_time/1000; 							/* convert milliseconds to seconds */

    hipEventDestroy(start);							
    hipEventDestroy(stop);
    hipFree(A_dev);									/* release gpu array */

}	


//--------------------------------------------------------------------------------------------------------------

/* one cell per thread, without using shared memory */
__global__ void kernel1(int k, int n, float *A) 
{

	/* same as Floyd Warshal Method but with the use of gpu */
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if (A[i*n+j] > A[i*n+k] + A[k*n+j]){
		A[i*n+j] = A[i*n+k] + A[k*n+j];
	}
}

//--------------------------------------------------------------------------------------------------------------

/* one cell per thread, using shared memory */
__global__ void kernel2(int k, int n, float *A) 
{

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	__shared__ float shA[1];

	shA[threadIdx.x]=A[i*n+k];	/* transfer A-array cell to shared mem cell */

	/* do the check using the shared memory cell */
	if (A[i*n+j] > shA[threadIdx.x] + A[k*n+j]) {
		A[i*n+j] = shA[threadIdx.x] + A[k*n+j];
	}

}

//--------------------------------------------------------------------------------------------------------------

/* multiple cellsPerThread (4) per thread, using shared memory */
__global__ void kernel3(int k, int n, float *A, int cellsPerThread) 
{

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int c, l, m;

	 __shared__ float shA[4];
	 
	/* transfering 4 A-array cells to shared mem cells */ 
	c = 0; 
	for (l=i*cellsPerThread; l<(i*cellsPerThread) + cellsPerThread; l++) {
		shA[threadIdx.x*cellsPerThread + c] = A[l*n + k];
		c++;
	}
	c = 0;
	
	__syncthreads();
	
	/* do the check using the shared memory cells */
	for (l=i*cellsPerThread; l<(i*cellsPerThread) + cellsPerThread; l++) {
		for (m=j*cellsPerThread; m<(j*cellsPerThread) + cellsPerThread; m++) {
			if (A[l*n + m] > shA[threadIdx.x*cellsPerThread + c] + A[k*n + m]) {
				A[l*n + m] = shA[threadIdx.x*cellsPerThread + c] + A[k*n + m];
			}
		}
		c++;
	}
}

//--------------------------------------------------------------------------------------------------------------

/* test cuda results with serial Floyd Warshall method */ 
void test(float *A) {
	int i;

	for (i=0; i<n*n; i++) {
		if (D[i] != A[i]) {
			printf("%sFailed!\n%s", KRED, RESET);
			return;
		}
	}
	
	printf("%sPassed!\n%s",KGRN ,RESET);
}


//--------------------------------------------------------------------------------------------------------------
