#include "hip/hip_runtime.h"
/* 		Hadjithoma Abdreas
 * 		Aristotle University of Thessaloniki		
 *		Sept 2016 		
 *		All Pair Shortest Path (APSP) Solution
 */



#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define RESET "\033[0m"

#define maxThreadsPerBlock 64 //8*8

void makeAdjacency();
void floydWarshallMethod();
void methods(int f, int t);
void test(float *A_dev);
void printArray(float *array);
void transposeAdjacency();
void cwf3();

__global__ void kernel1(int k, int n, float *A);
__global__ void kernel2(int k, int n, float *A);
__global__ void kernel3(int k, int n, float *A, int cellsPerThread);

__global__ void kernel1_t(int k, int n, float *A, float *AT);
__global__ void kernel2_t(int k, int n, float *A, float *AT);
__global__ void kernel3_t(int k, int n, float *A, float *AT, int cellsPerThread);

__global__ void kernel1_t_v2(int k, int n, float *A, float *AT);
__global__ void kernel2_t_v2(int k, int n, float *A, float *AT);
__global__ void kernel3_t_v2(int k, int n, float *A, float *AT, int cellsPerThread);



int n, w;		
float p;
float *D; 		/* D: distance (for the Floyd Warshall algorithm) */
float *A; 		/* A: array (for methods 1, 2, 3 as mentioned in the exercise) */
float *AT; 		/* A_t: transposed adjancecy array */
float *A_result11, *A_result21, *A_result31;	/* arrays for saving results in methods 1, 2, 3 */
float *A_result12, *A_result22, *A_result32;	/* arrays for saving results in methods 1, 2, 3 */
float *A_result13, *A_result23, *A_result33;	/* arrays for saving results in methods 1, 2, 3 */

float seq_time;
struct timeval startwtime, endwtime;
//hipEvent_t start, stop;\    



//--------------------------------------------------------------------------------------------------------------

int main(int argc, char** argv)
{

	if (argc!=4){
		printf("Enter: ./<name> n p w\n");
		exit(3);
	}

	n = 1<<atoi(argv[1]);
	p = atof(argv[2]);
	w = atoi(argv[3]);
   	printf("n = %d \np = %.2f \nw = %d\n", n, p, w);


    time_t t;
    srand((unsigned)time(&t));

    D = (float *)malloc(n*n*sizeof(float)); 
    A = (float *)malloc(n*n*sizeof(float)); 
    AT = (float *)malloc(n*n*sizeof(float)); 


	makeAdjacency(); 		/* initialize the graph */
	transposeAdjacency();	/* transpose the Adjency*/

	floydWarshallMethod();
	printf("Floyd Warshall time:\t%f\n", seq_time);


	/* init the timers */
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
 	

	//without A-Transpose
	//1
	printf("\n");
 	hipEventRecord(start);
	methods(1,1);
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),
	printf("CUDA Method 1 time:\t%f\t -> ", seq_time/1000);
	test(A_result11);
	free(A_result11);


	//2
	hipEventRecord(start);
	methods(2,1);
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),
	printf("CUDA Method 2 time:\t%f\t -> ", seq_time/1000);
    test(A_result21);
    free(A_result21);

    //3
    hipEventRecord(start);
    methods(3,1);
    hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),
	printf("CUDA Method 3 time:\t%f\t -> ", seq_time/1000);
    test(A_result31);
	free(A_result31);


	//with A transpose 
	//1
	printf("\n");
	hipEventRecord(start);
	methods(1,2);
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),	
  	printf("CUDA Method 1T time:\t%f\t -> ", seq_time/1000);
	test(A_result12);
	free(A_result12);

	//2
    hipEventRecord(start);
    methods(2,2);
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),	
  	printf("CUDA Method 2T time:\t%f\t -> ", seq_time/1000);
    test(A_result22);
    free(A_result22);	
	
	//3
    hipEventRecord(start);
	methods(3,2);
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),
  	printf("CUDA Method 3T time:\t%f\t -> ", seq_time/1000);
    test(A_result32);
	free(A_result32);

	//with A-transpose - version2
	//1
	printf("\n");
 	hipEventRecord(start);
	methods(1,3);
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),
	printf("CUDA Method 1T_v2 time:\t%f\t -> ", seq_time/1000);
	test(A_result13);
	free(A_result13);

	//2
 	hipEventRecord(start);
	methods(2,3);
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),
	printf("CUDA Method 2T_v2 time:\t%f\t -> ", seq_time/1000);
	test(A_result23);
	free(A_result23);

	//3
 	hipEventRecord(start);
	methods(3,3);
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&seq_time, start, stop),
	printf("CUDA Method 3T_v2 time:\t%f\t -> ", seq_time/1000);
	test(A_result33);
	free(A_result33);

	free(D);
	free(A);
    return 1;
}

//--------------------------------------------------------------------------------------------------------------

/* create the array that represents the graph */
void makeAdjacency()
{
	int i, j;
	float randValue;

	for (i=0; i<n; i++){
		for (j=0; j<n; j++){
			randValue = (((double)rand()+1)/((double)RAND_MAX+1)); /* Get a random float value from 0 to 1 */
			if (randValue>p)
				A[i*n+j] = INFINITY;
			else 
				A[i*n+j] = randValue * w;
		}
        A[i*n+i] = 0;
	}
}

//--------------------------------------------------------------------------------------------------------------

void transposeAdjacency()
{
	int i, j;
	for (i=0; i<n; i++){
		for (j=0; j<n; j++){
			AT[i*n+j] = A[j*n+i];
		}
	}
}


//--------------------------------------------------------------------------------------------------------------
/* print array for personal check (only for small n) */
void printArray(float *array) {
    int i;
    
    if (n>3*3) return; 

    for (i=0; i<n*n; i++) {
    	if (i % n == 0) printf("\n"); /* change line */
    	printf("%f  ",array[i]);
    }

    printf("\n");	
}

//--------------------------------------------------------------------------------------------------------------

/* floyd Warshall Method based on this: http://www.programming-algorithms.net/article/45708/Floyd-Warshall-algorithm */
void floydWarshallMethod()
{
	int i, j, k;

	/* copy arrays D <- A */
	for (i=0; i<n; i++){
		for (j=0; j<n; j++){
			D[i*n+j] = A[i*n+j];
		}
		D[i*n+i] = 0;
	}

    gettimeofday (&startwtime, NULL);

	for (k=0; k<n; k++){
		for (i=0; i<n; i++){
			for (j=0; j<n; j++){
                if (D[i*n+j] > D[i*n+k] + D[k*n+j]){
                    D[i*n+j] = D[i*n+k] + D[k*n+j];
                }
			}
		}
	}
	gettimeofday (&endwtime, NULL);
    seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
}


//--------------------------------------------------------------------------------------------------------------

/* Alling kernel functions for methods 1, 2, 3 and Alculating the execution time */
/* where f = 1, 2, 3 for methods 1, 2 or 3 respectively */
void methods(int f, int t)
{
	float *A_dev;
	float *AT_dev;												/* gpu array */
	int k;

	hipMalloc((void **)&A_dev, n*n*sizeof(float)); 			/* alloATe the memory in GPU */ 		
	if (t) hipMalloc((void **)&AT_dev, n*n*sizeof(float)); 	/* alloATe the memory in GPU */ 		

    /************************************************ 1st Method ************************************************/
	if (f==1 && t==1){
		A_result11 = (float *) malloc (n*n*sizeof(float));

		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */

		if (n*n<=maxThreadsPerBlock){ 										/* 64 threads (max) per block */
	     	dim3 threadsPerBlock(n, n);
	    	for (k=0; k<n; k++)
	    		kernel1<<<1, threadsPerBlock>>>(k, n, A_dev);
		}
		else{			/* if threads per block > 64 */
			dim3 threadsPerBlock(8,8);
			dim3 blocksPerGrid(n/threadsPerBlock.x, n/threadsPerBlock.y);
			for (k=0; k<n; k++)
				kernel1<<<blocksPerGrid, threadsPerBlock>>>(k, n, A_dev);
		}
	   
	   hipMemcpy(A_result11, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost); /* get results back to CPU */
	}

    /******************************************* 1st Method using A^T *******************************************/
	else if (f==1 && t==2){
		A_result12 = (float *) malloc (n*n*sizeof(float));

		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */
		hipMemcpy(AT_dev, AT, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table AΤ from CPU to GPU */


		if (n*n<=maxThreadsPerBlock){ 										/* 64 threads (max) per block */
	     	dim3 threadsPerBlock(n, n);
	    	for (k=0; k<n; k++)
	    		kernel1_t<<<1, threadsPerBlock>>>(k, n, A_dev, AT_dev);
		}
		else{			/* if threads per block > 64 */
			dim3 threadsPerBlock(8,8);
			dim3 blocksPerGrid(n/threadsPerBlock.x, n/threadsPerBlock.y);
			for (k=0; k<n; k++)
				kernel1_t<<<blocksPerGrid, threadsPerBlock>>>(k, n, A_dev, AT_dev);
		}
	   
	    hipMemcpy(A_result12, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost); /* get results back to CPU */
	}

	/*************************************** 1st Method using A^T - Ver.2 ***************************************/
	else if (f==1 && t==3){
		A_result13 = (float *) malloc (n*n*sizeof(float));

		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */
		hipMemcpy(AT_dev, AT, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table AT from CPU to GPU */

	    if (n*n<=maxThreadsPerBlock) {
	    	int threadsPerBlock =n*n; 
	    	dim3 blocksPerGrid((n+threadsPerBlock)/threadsPerBlock, n);
	    	
	    	for (int k=0; k<n; k++) {
				kernel1_t_v2<<<blocksPerGrid, threadsPerBlock>>>(k, n, A_dev, AT_dev);
	    	}
	    }
	    else {
	    	int threadsPerBlock = 128; 
	    	dim3 blocksPerGrid((n+threadsPerBlock)/threadsPerBlock, n);
	    	
	    	for (int k=0; k<n; k++) {
				kernel1_t_v2<<<blocksPerGrid, threadsPerBlock>>>(k, n, A_dev, AT_dev);
	    	}
		}	
	   	hipMemcpy(A_result13, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}


    /************************************************ 2nd Method ************************************************/
	/* same as 1st method but this time we are Alling different kernel function that uses shared mem */
	else if (f==2 && t==1){

		A_result21 = (float *) malloc (n*n*sizeof(float));
		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */

		if (n*n<=maxThreadsPerBlock){ 										/* 64 threads (max) per block */
	     	dim3 threadsPerBlock(n, n);
    		size_t size = threadsPerBlock.x*sizeof(float);
	    	for (k=0; k<n; k++)
	    		kernel2<<<1, threadsPerBlock, size>>>(k, n, A_dev);
		}
		else{
			dim3 threadsPerBlock(8, 8);
			dim3 blocksPerGrid(n/threadsPerBlock.x, n/threadsPerBlock.y);
    		size_t size = threadsPerBlock.x*sizeof(float);

			for (k=0; k<n; k++)
				kernel2<<<blocksPerGrid, threadsPerBlock, size>>>(k, n, A_dev);
		}
	   
	   	hipMemcpy(A_result21, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}

    /******************************************* 2nd Method using A^T *******************************************/
	else if (f==2 && t==2){
		A_result22 = (float *) malloc (n*n*sizeof(float));

		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */
		hipMemcpy(AT_dev, AT, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table AT from CPU to GPU */

		if (n*n<=maxThreadsPerBlock){ 										/* 64 threads (max) per block */
	     	dim3 threadsPerBlock(n, n);
	    	for (k=0; k<n; k++)
	    		kernel2_t<<<1, threadsPerBlock>>>(k, n, A_dev, AT_dev);
		}
		else{
			dim3 threadsPerBlock(8,8);
			dim3 blocksPerGrid(n/threadsPerBlock.x, n/threadsPerBlock.y);
			for (k=0; k<n; k++)
				kernel2_t<<<blocksPerGrid, threadsPerBlock>>>(k, n, A_dev, AT_dev);
		}
	   
	   	hipMemcpy(A_result22, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}

	/*************************************** 2nd Method using A^T - Ver.2 ***************************************/
	else if (f==2 && t==3){
		A_result23 = (float *) malloc (n*n*sizeof(float));

		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */
		hipMemcpy(AT_dev, AT, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table AT from CPU to GPU */

	    if (n*n<=maxThreadsPerBlock) {
	    	int threadsPerBlock =n*n; 
	    	dim3 blocksPerGrid((n+threadsPerBlock)/threadsPerBlock, n);
	    	
	    	for (int k=0; k<n; k++) {
				kernel2_t_v2<<<blocksPerGrid,threadsPerBlock>>>(k, n, A_dev, AT_dev);
	    	}
	    }
	    else {
	    	int threadsPerBlock = 256; 
	    	dim3 blocksPerGrid((n+threadsPerBlock)/threadsPerBlock, n);
	    	
	    	for (int k=0; k<n; k++) {
				kernel2_t_v2<<<blocksPerGrid,threadsPerBlock>>>(k, n, A_dev, AT_dev);
	    	}
		}	
	   	hipMemcpy(A_result23, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}


    /************************************************ 3rd Method ************************************************/
	/* same as 2nd method but based on 4 cells per thread */
	else if (f==3 && t==1){
		A_result31 = (float *) malloc (n*n*sizeof(float));
		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */

		int cellsPerThread = 4;												/* number of cells per thread */

		if (n*n<=maxThreadsPerBlock * cellsPerThread){ 
   	    	dim3 threadsPerBlock(n/cellsPerThread, n/cellsPerThread);
   	    	size_t size = (threadsPerBlock.x*cellsPerThread)*sizeof(float) + (threadsPerBlock.y*cellsPerThread)*sizeof(float);
			for (k=0; k<n; k++)
				kernel3<<<1, threadsPerBlock, size>>>(k, n, A_dev, cellsPerThread);
		}
		else{
			dim3 threadsPerBlock(8,8);
    		dim3 blocksPerGrid(n/(threadsPerBlock.x * cellsPerThread), n/(threadsPerBlock.y * cellsPerThread));
    		size_t size = (threadsPerBlock.x*cellsPerThread)*sizeof(float) + (threadsPerBlock.y*cellsPerThread)*sizeof(float);
			for (k=0; k<n; k++)
				kernel3<<<blocksPerGrid, threadsPerBlock, size>>>(k, n, A_dev, cellsPerThread);

		}
	    hipMemcpy(A_result31, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}

    /******************************************* 3rd Method using A^T *******************************************/
	else if (f==3 && t==2){
		A_result32 = (float *) malloc (n*n*sizeof(float));
		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */
		hipMemcpy(AT_dev, AT, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table AT from CPU to GPU */

		int cellsPerThread = 4;												/* number of cells per thread */

		if (n<= 8 * cellsPerThread){ 
   	    	dim3 threadsPerBlock(n/cellsPerThread, n/cellsPerThread);
   	    	size_t size = (threadsPerBlock.x*cellsPerThread)*sizeof(float) + (threadsPerBlock.y*cellsPerThread)*sizeof(float);
			for (k=0; k<n; k++)
				kernel3_t<<<1, threadsPerBlock, size>>>(k, n, A_dev, AT_dev, cellsPerThread);
		}

		else{
			dim3 threadsPerBlock(8,8);
    		dim3 blocksPerGrid(n/(threadsPerBlock.x * cellsPerThread), n/(threadsPerBlock.y * cellsPerThread));
    		size_t size = (threadsPerBlock.x*cellsPerThread)*sizeof(float) + (threadsPerBlock.y*cellsPerThread)*sizeof(float);
			
			for (k=0; k<n; k++)
				kernel3_t<<<blocksPerGrid, threadsPerBlock, size>>>(k, n, A_dev, AT_dev, cellsPerThread);

		}
	    hipMemcpy(A_result32, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}

	/*************************************** 3rd Method using A^T - Ver.2 ***************************************/
	else if (f==3 && t==3){
		A_result33 = (float *) malloc (n*n*sizeof(float));

		hipMemcpy(A_dev, A, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table A from CPU to GPU */
		hipMemcpy(AT_dev, AT, n*n*sizeof(float), hipMemcpyHostToDevice);	/* copy table AT from CPU to GPU */
		int cellsPerThread = 4;												/* number of cells per thread */

    	int threadsPerBlock = n/cellsPerThread; 
    	dim3 blocksPerGrid((n+threadsPerBlock)/(threadsPerBlock*cellsPerThread), n/cellsPerThread);
    	size_t size = 2*threadsPerBlock*cellsPerThread*sizeof(float);
    	
    	for (int k=0; k<n; k++) {
			kernel3_t_v2<<<blocksPerGrid, threadsPerBlock, size>>>(k, n, A_dev, AT_dev, cellsPerThread);;
    	}
	    
	   	hipMemcpy(A_result33, A_dev, n*n*sizeof(float), hipMemcpyDeviceToHost);
	}
    hipFree(AT_dev);
}	
	

//--------------------------------------------------------------------------------------------------------------

/* one cell per thread, without using shared memory */
__global__ void kernel1(int k, int n, float *A) 
{

	/* same as Floyd Warshal Method but with the use of gpu */
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
		
	if (A[i*n+j] > A[i*n+k] + A[k*n+j]){
		A[i*n+j] = A[i*n+k] + A[k*n+j];
	}
}

//--------------------------------------------------------------------------------------------------------------


/* method 1 using the transpose of A */
__global__ void kernel1_t(int k, int n, float *A, float *AT) 
{

	/* same as Floyd Warshal Method but with the use of gpu */
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if (A[i*n+j] > AT[k*n+i] + A[k*n+j]){
		A[i*n+j] = AT[n*j+i] = AT[k*n+i] + A[k*n+j];
	} 

}

//--------------------------------------------------------------------------------------------------------------

/* one cell per thread, using shared memory */
__global__ void kernel2(int k, int n, float *A) 
{

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	extern __shared__ float shA[];		

	shA[threadIdx.x]=A[i*n+k];	/* transfer A-array cell to shared mem cell */
	/* do the check using the shared memory cell */	  	

	if (A[i*n+j] > shA[threadIdx.x] + A[k*n+j]) {
		A[i*n+j] = shA[threadIdx.x] + A[k*n+j];
	}
}

//--------------------------------------------------------------------------------------------------------------


/* method 2 using the transpose of A */
__global__ void kernel2_t(int k, int n, float *A, float *AT) 
{

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	__shared__ float shA[1];

	shA[threadIdx.x]=AT[n*k+i];	

	if (A[i*n+j] > shA[threadIdx.x] + A[k*n+j]) {
		A[i*n+j] = AT[j*n+i] = shA[threadIdx.x] + A[k*n+j];
	}
}

//--------------------------------------------------------------------------------------------------------------

/* multiple cellsPerThread (4) per thread, using shared memory */
__global__ void kernel3(int k, int n, float *A, int cellsPerThread) 
{

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int c, l, m;

	extern __shared__ float shared[];
	float *shA = (float *)&shared[blockDim.x*cellsPerThread]; 

	/* transfering 4 A-array cells to shared mem cells */ 
	c = 0; 
	for (l=i*cellsPerThread; l<(i*cellsPerThread) + cellsPerThread; l++) {
		shA[threadIdx.x*cellsPerThread + c] = A[l*n+k];
		c++;
	}
	c = 0;
	
	__syncthreads();
	
	/* do the check using the shared memory cells */
	for (l=i*cellsPerThread; l<(i*cellsPerThread) + cellsPerThread; l++) {
		for (m=j*cellsPerThread; m<(j*cellsPerThread) + cellsPerThread; m++) {
			if (A[l*n+m] > shA[threadIdx.x*cellsPerThread + c] + A[k*n+m]) {
				A[l*n+m] = shA[threadIdx.x*cellsPerThread + c] + A[k*n+m];
			}
		}
		c++;
	}
}

//--------------------------------------------------------------------------------------------------------------


/* method 3 using the transpose of A */
__global__ void kernel3_t(int k, int n, float *A, float *AT, int cellsPerThread) 
{


	int li,lj,l, m;
	int ci=0, cj=0;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.x + threadIdx.y;
	
	extern __shared__ float shared[];
	float *shA1 = shared;
	float *shA2 = (float *)&shA1[blockDim.x*cellsPerThread];
	
	for (li=i*cellsPerThread; li<(i*cellsPerThread) + cellsPerThread; li++) {
		shA1[threadIdx.x*cellsPerThread + ci] = AT[k*n + li];
		ci++;
	}
	ci = 0;
	
	for (lj=j*cellsPerThread; lj<(j*cellsPerThread) + cellsPerThread; lj++) {
		shA2[threadIdx.y*cellsPerThread + cj] = A[k*n + lj];
		cj++;
	}
	cj = 0;
	
	__syncthreads();
	
	for (l=i*cellsPerThread; l<(i*cellsPerThread) + cellsPerThread; l++) {
		for (m=j*cellsPerThread; m<(j*cellsPerThread) + cellsPerThread; m++) {
			if ( A[l*n + m] > shA1[threadIdx.x*cellsPerThread + ci] + shA2[threadIdx.y*cellsPerThread + cj] ) {
				A[l*n + m] = AT[m*n + l] = shA1[threadIdx.x*cellsPerThread + ci] + shA2[threadIdx.y*cellsPerThread + cj];
			}
			cj++;
		}
		cj = 0;
		ci++;
	}
}

//--------------------------------------------------------------------------------------------------------------

__global__ void kernel1_t_v2(int k, int n, float *A, float *AT)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i >= n) 
		return;
	
	if (A[n*blockIdx.y + i] > AT[n*k + blockIdx.y] + A[k*n + i] ){
		A[n*blockIdx.y + i] = AT[i*n + blockIdx.y] = AT[n*k + blockIdx.y] + A[k*n + i];
	}
	
}

//--------------------------------------------------------------------------------------------------------------

__global__ void kernel2_t_v2(int k, int n, float *A, float *AT)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i >= n) 
		return;

	//use shared mem
	__shared__ float shA;	
	shA = AT[n*k + blockIdx.y];

	if (A[n*blockIdx.y + i] > shA + A[k*n + i])
		A[n*blockIdx.y + i] = AT[i*n + blockIdx.y] = shA + A[k*n + i];
}

//--------------------------------------------------------------------------------------------------------------

__global__ void kernel3_t_v2(int k, int n, float *A, float *AT, int cellsPerThread)
{
	int li, lj, l, m;
	int ci = 0, cj = 0;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i >= n) return;
	
	extern __shared__ float s[];
	float *shA1 = s;
	float *shA2 = (float *)&shA1[blockDim.x*cellsPerThread];
	
	for (li=blockIdx.y*cellsPerThread; li<(blockIdx.y*cellsPerThread) + cellsPerThread; li++) {
		shA1[threadIdx.x*cellsPerThread + ci] = AT[k*n + li];
		ci++;
	}
	ci = 0;
	
	for (lj=i*cellsPerThread; lj<(i*cellsPerThread) + cellsPerThread; lj++) {
		shA2[threadIdx.x*cellsPerThread + cj] = A[k*n + lj];
		cj++;
	}
	cj = 0;
	
	__syncthreads();

	for (l=blockIdx.y*cellsPerThread; l<(blockIdx.y*cellsPerThread) + cellsPerThread; l++) {
		for (m=i*cellsPerThread; m<(i*cellsPerThread) + cellsPerThread; m++) {
			if (A[l*n + m] > shA1[threadIdx.x*cellsPerThread + ci] + shA2[threadIdx.x*cellsPerThread + cj] ) {
				A[l*n + m] = AT[m*n + l] = shA1[threadIdx.x*cellsPerThread + ci] + shA2[threadIdx.x*cellsPerThread + cj];
			}
			cj++;
		}
		cj = 0;
		ci++;
	}
}

//--------------------------------------------------------------------------------------------------------------

/* test cuda results with serial Floyd Warshall method */ 
void test(float *A) {
	int i;

	for (i=0; i<n*n; i++) {
		if (D[i] != A[i]) {
			printf("%sFailed!\n%s", KRED, RESET);
			return;
		}
	}
	
	printf("%sPassed!\n%s",KGRN ,RESET);
}

//--------------------------------------------------------------------------------------------------------------

